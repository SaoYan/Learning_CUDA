#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

void initialData(float *ip, const int size);
float naiveReduce(float *data, int size);
float neighboredPairReduce(float *data, const int size);
float interleavedPairReduce(float *data, const int size);

__global__ void reduceNeighbored(float *g_idata, float * g_odata, const int n);
__global__ void reduceNeighboredLessDiv(float *g_idata, float * g_odata, const int n);
__global__ void reduceInterleaved(float *g_idata, float * g_odata, const int n);

__global__ void reduceUnrolling2(float *g_idata, float * g_odata, const int n);
__global__ void reduceUnrolling4(float *g_idata, float * g_odata, const int n);
__global__ void reduceUnrolling8(float *g_idata, float * g_odata, const int n);
__global__ void reduceUnrolling16(float *g_idata, float * g_odata, const int n);

__global__ void reduceUnrollWarps8(float *g_idata, float * g_odata, const int n);
__global__ void reduceCompleteUnrollWarps8(float *g_idata, float * g_odata, const int n);

template <int iBlockSize>
__global__ void reduceCompleteUnroll(float *g_idata, float * g_odata, const int n);

#define CHECK(call) {                                                        \
    const hipError_t error = call;                                          \
    if (error != hipSuccess) {                                              \
        printf("Error: %s:%d, ", __FILE__, __LINE__);                        \
        printf("code: %d, reason: %s\n", error, hipGetErrorString(error));  \
        exit(1);                                                             \
    }                                                                        \
}                                                                            \

int main(int argc, char **argv) {
    int size = 1<<24, evenSize = size;
    if (evenSize % 2 != 0) evenSize++; // should be even for pair-reducution to work
    printf("Vector size %d %d\n", size, evenSize);

    size_t nBytes = evenSize * sizeof(float);
    clock_t start, end;
    double exeTime;
    float reductionSum;

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp; 
    CHECK(hipGetDeviceProperties(&deviceProp, dev)); 
    printf("Using Device %d: %s\n", dev, deviceProp.name); 
    CHECK(hipSetDevice(dev));

    // grid and block configuration
    int blockSize = 512;
    if (argc > 1) { // support input arg from terminal
        blockSize = atoi(argv[1]);
    }
    dim3 block(blockSize);
    dim3 grid((evenSize + block.x - 1)/ block.x);
    printf("Grid dimension %d Block dimensiton %d\n", grid.x, block.x);

    // allocate host memory
    float *h_idata, *h_odata, *h_idata_cpy;
    h_idata = (float *) malloc(nBytes);
    h_odata = (float *) malloc(grid.x * sizeof(float));
    h_idata_cpy = (float *) malloc(nBytes);
    initialData(h_idata, size);
    memcpy(h_idata_cpy, h_idata, nBytes);

    // 0. compute on CPU
    start = clock();
    reductionSum = naiveReduce(h_idata_cpy, size);
    // reductionSum = neighboredPairReduce(h_idata_cpy, evenSize);
    // reductionSum = interleavedPairReduce(h_idata_cpy, evenSize);
    end = clock();
    exeTime = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("\nCPU reduce:                   execution time %.4f ms, result %f\n\n", exeTime * 1e3, reductionSum);

    // allocate device memory
    float *d_idata, *d_odata;
    CHECK(hipMalloc((float**)&d_idata, nBytes));
    CHECK(hipMalloc((float**)&d_odata, grid.x * sizeof(float)));

    // just warm up
    CHECK(hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice));
    reduceNeighbored<<<grid, block>>>(d_idata, d_odata, evenSize);
    CHECK(hipDeviceSynchronize());

    // 1. GPU - neighbored pair reduce
    CHECK(hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice));
    start = clock();
    // CUDA part
    reduceNeighbored<<<grid, block>>>(d_idata, d_odata, evenSize);
    CHECK(hipDeviceSynchronize());
    end = clock();
    exeTime = ((double) (end - start)) / CLOCKS_PER_SEC;
    // Host part
    reductionSum = 0.0f;
    CHECK(hipMemcpy(h_odata, d_odata, grid.x * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < grid.x; i++) reductionSum += h_odata[i];
    printf("GPU neighbored pair reduce:   execution time %.4f ms, result %f\n", exeTime * 1e3, reductionSum);

    // 2. GPU - neighbored pair reduce 2
    CHECK(hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice));
    start = clock();
    // CUDA part
    reduceNeighboredLessDiv<<<grid, block>>>(d_idata, d_odata, evenSize);
    CHECK(hipDeviceSynchronize());
    end = clock();
    exeTime = ((double) (end - start)) / CLOCKS_PER_SEC;
    // Host part
    reductionSum = 0.0f;
    CHECK(hipMemcpy(h_odata, d_odata, grid.x * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < grid.x; i++) reductionSum += h_odata[i];
    printf("GPU neighbored pair reduce 2: execution time %.4f ms, result %f\n\n", exeTime * 1e3, reductionSum);

    // 3. GPU - interleaved pair reduce
    CHECK(hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice));
    start = clock();
    // CUDA part
    reduceInterleaved<<<grid, block>>>(d_idata, d_odata, evenSize);
    CHECK(hipDeviceSynchronize());
    end = clock();
    exeTime = ((double) (end - start)) / CLOCKS_PER_SEC;
    // Host part
    reductionSum = 0.0f;
    CHECK(hipMemcpy(h_odata, d_odata, grid.x * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < grid.x; i++) reductionSum += h_odata[i];
    printf("GPU interleaved pair reduce:  execution time %.4f ms, result %f\n", exeTime * 1e3, reductionSum);

    // 4. GPU - interleaved pair reduce; x2 unrolling
    CHECK(hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice));
    start = clock();
    // CUDA part
    reduceUnrolling2<<<grid.x / 2, block>>>(d_idata, d_odata, evenSize);
    CHECK(hipDeviceSynchronize());
    end = clock();
    exeTime = ((double) (end - start)) / CLOCKS_PER_SEC;
    // Host part
    reductionSum = 0.0f;
    CHECK(hipMemcpy(h_odata, d_odata, grid.x / 2 * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < grid.x / 2; i++) reductionSum += h_odata[i];
    printf("GPU x2 unrolling:             execution time %.4f ms, result %f\n", exeTime * 1e3, reductionSum);

    // 5. GPU - interleaved pair reduce; x4 unrolling
    CHECK(hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice));
    start = clock();
    // CUDA part
    reduceUnrolling4<<<grid.x / 4, block>>>(d_idata, d_odata, evenSize);
    CHECK(hipDeviceSynchronize());
    end = clock();
    exeTime = ((double) (end - start)) / CLOCKS_PER_SEC;
    // Host part
    reductionSum = 0.0f;
    CHECK(hipMemcpy(h_odata, d_odata, grid.x / 4 * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < grid.x / 4; i++) reductionSum += h_odata[i];
    printf("GPU x4 unrolling:             execution time %.4f ms, result %f\n", exeTime * 1e3, reductionSum);

    // 6. GPU - interleaved pair reduce; x8 unrolling
    CHECK(hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice));
    start = clock();
    // CUDA part
    reduceUnrolling8<<<grid.x / 8, block>>>(d_idata, d_odata, evenSize);
    CHECK(hipDeviceSynchronize());
    end = clock();
    exeTime = ((double) (end - start)) / CLOCKS_PER_SEC;
    // Host part
    reductionSum = 0.0f;
    CHECK(hipMemcpy(h_odata, d_odata, grid.x / 8 * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < grid.x / 8; i++) reductionSum += h_odata[i];
    printf("GPU x8 unrolling:             execution time %.4f ms, result %f\n", exeTime * 1e3, reductionSum);

    // 6.5. GPU - interleaved pair reduce; x16 unrolling
    CHECK(hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice));
    start = clock();
    // CUDA part
    reduceUnrolling16<<<grid.x / 16, block>>>(d_idata, d_odata, evenSize);
    CHECK(hipDeviceSynchronize());
    end = clock();
    exeTime = ((double) (end - start)) / CLOCKS_PER_SEC;
    // Host part
    reductionSum = 0.0f;
    CHECK(hipMemcpy(h_odata, d_odata, grid.x / 16 * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < grid.x / 16; i++) reductionSum += h_odata[i];
    printf("GPU x16 unrolling:            execution time %.4f ms, result %f\n\n", exeTime * 1e3, reductionSum);

    // 7. GPU - unrolling warps
    CHECK(hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice));
    start = clock();
    // CUDA part
    reduceUnrollWarps8<<<grid.x / 8, block>>>(d_idata, d_odata, evenSize);
    CHECK(hipDeviceSynchronize());
    end = clock();
    exeTime = ((double) (end - start)) / CLOCKS_PER_SEC;
    // Host part
    reductionSum = 0.0f;
    CHECK(hipMemcpy(h_odata, d_odata, grid.x / 8 * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < grid.x / 8; i++) reductionSum += h_odata[i];
    printf("GPU unrolling warps:          execution time %.4f ms, result %f\n", exeTime * 1e3, reductionSum);

    // 8. GPU - complete unrolling 
    CHECK(hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice));
    start = clock();
    // CUDA part
    reduceCompleteUnrollWarps8<<<grid.x / 8, block>>>(d_idata, d_odata, evenSize);
    CHECK(hipDeviceSynchronize());
    end = clock();
    exeTime = ((double) (end - start)) / CLOCKS_PER_SEC;
    // Host part
    reductionSum = 0.0f;
    CHECK(hipMemcpy(h_odata, d_odata, grid.x / 8 * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < grid.x / 8; i++) reductionSum += h_odata[i];
    printf("GPU complete unrolling:       execution time %.4f ms, result %f\n\n", exeTime * 1e3, reductionSum);

    // 9. GPU - template function 
    CHECK(hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice));
    start = clock();
    // CUDA part
    switch (blockSize) {
        case 1024:
            reduceCompleteUnroll<1024><<<grid.x / 8, block>>>(d_idata, d_odata, evenSize);
            break;
        case 512:
            reduceCompleteUnroll<512><<<grid.x / 8, block>>>(d_idata, d_odata, evenSize);
            break;
        case 256:
            reduceCompleteUnroll<256><<<grid.x / 8, block>>>(d_idata, d_odata, evenSize);
            break;
        case 128:
            reduceCompleteUnroll<128><<<grid.x / 8, block>>>(d_idata, d_odata, evenSize);
            break;
        case 64:
            reduceCompleteUnroll<64><<<grid.x / 8, block>>>(d_idata, d_odata, evenSize);
            break;
    }
    CHECK(hipDeviceSynchronize());
    end = clock();
    exeTime = ((double) (end - start)) / CLOCKS_PER_SEC;
    // Host part
    reductionSum = 0.0f;
    CHECK(hipMemcpy(h_odata, d_odata, grid.x / 8 * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < grid.x / 8; i++) reductionSum += h_odata[i];
    printf("GPU template function:        execution time %.4f ms, result %f\n", exeTime * 1e3, reductionSum);

    // free host mem
    free(h_idata);
    free(h_odata);
    free(h_idata_cpy);

    // free device mem
    CHECK(hipFree(d_idata));
    CHECK(hipFree(d_odata));
    
    // clean up all resources
    CHECK(hipDeviceReset());
    return 0;
}

/**********CUDA kernels**********/

__global__ void reduceNeighbored(float *g_idata, float * g_odata, const int n) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    const int tid = threadIdx.x;

    // convert global data pointer to the local pointer of this block
    float *idata = g_idata + blockIdx.x * blockDim.x;

    // in-place reduction in global memory
    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        if (tid % (stride * 2) == 0) {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}


__global__ void reduceNeighboredLessDiv(float *g_idata, float * g_odata, const int n) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    const int tid = threadIdx.x;

    // convert global data pointer to the local pointer of this block
    float *idata = g_idata + blockIdx.x * blockDim.x;

    // in-place reduction in global memory
    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        int i = tid * stride * 2;
        if (i < blockDim.x) {
            idata[i] += idata[i + stride];
        }
        __syncthreads();
    }

    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}


__global__ void reduceInterleaved(float *g_idata, float * g_odata, const int n) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    const int tid = threadIdx.x;

    // convert global data pointer to the local pointer of this block
    float *idata = g_idata + blockIdx.x * blockDim.x;

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads(); 
    }

    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}


__global__ void reduceUnrolling2(float *g_idata, float * g_odata, const int n) {
    const int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    if (idx >= n) return;
    const int tid = threadIdx.x;

    // convert global data pointer to the local pointer of this block
    float *idata = g_idata + blockIdx.x * blockDim.x * 2;

    // unrolling 2 data blocks
    if (idx + blockDim.x < n) g_idata[idx] += g_idata[idx + blockDim.x];
    __syncthreads();

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads(); 
    }

    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}


__global__ void reduceUnrolling4(float *g_idata, float * g_odata, const int n) {
    const int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;
    if (idx >= n) return;
    const int tid = threadIdx.x;

    // convert global data pointer to the local pointer of this block
    float *idata = g_idata + blockIdx.x * blockDim.x * 4;

    // unrolling 4
    if (idx + 3 * blockDim.x < n){
        float a1 = g_idata[idx];
        float a2 = g_idata[idx + blockDim.x];
        float a3 = g_idata[idx + 2 * blockDim.x];
        float a4 = g_idata[idx + 3 * blockDim.x];
        g_idata[idx] = a1 + a2 + a3 + a4;
    }
    __syncthreads();

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads(); 
    }

    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}


__global__ void reduceUnrolling8(float *g_idata, float * g_odata, const int n) {
    const int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;
    if (idx >= n) return;
    const int tid = threadIdx.x;

    // convert global data pointer to the local pointer of this block
    float *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // unrolling 8
    if (idx + 7 * blockDim.x < n) {
        float a1 = g_idata[idx];
        float a2 = g_idata[idx + blockDim.x];
        float a3 = g_idata[idx + 2 * blockDim.x];
        float a4 = g_idata[idx + 3 * blockDim.x];
        float a5 = g_idata[idx + 4 * blockDim.x];
        float a6 = g_idata[idx + 5 * blockDim.x];
        float a7 = g_idata[idx + 6 * blockDim.x];
        float a8 = g_idata[idx + 7 * blockDim.x];
        g_idata[idx] = a1 + a2 + a3 + a4 + a5 + a6 + a7 + a8;
    }
    __syncthreads();

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads(); 
    }

    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceUnrolling16(float *g_idata, float * g_odata, const int n) {
    const int idx = blockIdx.x * blockDim.x * 16 + threadIdx.x;
    if (idx >= n) return;
    const int tid = threadIdx.x;

    // convert global data pointer to the local pointer of this block
    float *idata = g_idata + blockIdx.x * blockDim.x * 16;

    // unrolling 16
    if (idx + 15 * blockDim.x < n) {
        float a1 = g_idata[idx];
        float a2 = g_idata[idx + blockDim.x];
        float a3 = g_idata[idx + 2 * blockDim.x];
        float a4 = g_idata[idx + 3 * blockDim.x];
        float a5 = g_idata[idx + 4 * blockDim.x];
        float a6 = g_idata[idx + 5 * blockDim.x];
        float a7 = g_idata[idx + 6 * blockDim.x];
        float a8 = g_idata[idx + 7 * blockDim.x];
        float a9 = g_idata[idx + 8 * blockDim.x];
        float a10 = g_idata[idx + 9 * blockDim.x];
        float a11 = g_idata[idx + 10 * blockDim.x];
        float a12 = g_idata[idx + 11 * blockDim.x];
        float a13 = g_idata[idx + 12 * blockDim.x];
        float a14 = g_idata[idx + 13 * blockDim.x];
        float a15 = g_idata[idx + 14 * blockDim.x];
        float a16 = g_idata[idx + 15 * blockDim.x];
        g_idata[idx] = a1 + a2 + a3 + a4 + a5 + a6 + a7 + a8 + 
            a9 + a10 + a11 + a12 + a13 + a14 + a15 + a16;
    }
    __syncthreads();

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads(); 
    }

    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceUnrollWarps8(float *g_idata, float * g_odata, const int n) {
    const int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;
    if (idx >= n) return;
    const int tid = threadIdx.x;

    // convert global data pointer to the local pointer of this block
    float *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // unrolling 8
    if (idx + 7 * blockDim.x < n) {
        float a1 = g_idata[idx];
        float a2 = g_idata[idx + blockDim.x];
        float a3 = g_idata[idx + 2 * blockDim.x];
        float a4 = g_idata[idx + 3 * blockDim.x];
        float a5 = g_idata[idx + 4 * blockDim.x];
        float a6 = g_idata[idx + 5 * blockDim.x];
        float a7 = g_idata[idx + 6 * blockDim.x];
        float a8 = g_idata[idx + 7 * blockDim.x];
        g_idata[idx] = a1 + a2 + a3 + a4 + a5 + a6 + a7 + a8;

        // float a1 = idata[tid];
        // float a2 = idata[tid + blockDim.x];
        // float a3 = idata[tid + 2 * blockDim.x];
        // float a4 = idata[tid + 3 * blockDim.x];
        // float a5 = idata[tid + 4 * blockDim.x];
        // float a6 = idata[tid + 5 * blockDim.x];
        // float a7 = idata[tid + 6 * blockDim.x];
        // float a8 = idata[tid + 7 * blockDim.x];
        // idata[tid] = a1 + a2 + a3 + a4 + a5 + a6 + a7 + a8;
    }
    __syncthreads();

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 32; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads(); 
    }

    // unrolling warp
    if (tid < 32) {
        volatile float *vmem = idata;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid +  8];
        vmem[tid] += vmem[tid +  4];
        vmem[tid] += vmem[tid +  2];
        vmem[tid] += vmem[tid +  1];
    }

    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceCompleteUnrollWarps8(float *g_idata, float * g_odata, const int n) {
    const int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;
    if (idx >= n) return;
    const int tid = threadIdx.x;

    // convert global data pointer to the local pointer of this block
    float *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // unrolling 8
    if (idx + 7 * blockDim.x < n) {
        float a1 = g_idata[idx];
        float a2 = g_idata[idx + blockDim.x];
        float a3 = g_idata[idx + 2 * blockDim.x];
        float a4 = g_idata[idx + 3 * blockDim.x];
        float a5 = g_idata[idx + 4 * blockDim.x];
        float a6 = g_idata[idx + 5 * blockDim.x];
        float a7 = g_idata[idx + 6 * blockDim.x];
        float a8 = g_idata[idx + 7 * blockDim.x];
        g_idata[idx] = a1 + a2 + a3 + a4 + a5 + a6 + a7 + a8;
    }
    __syncthreads();

    // in-place reduction and complete unroll
    if (blockDim.x >= 1024 && tid < 512) idata[tid] += idata[tid + 512];
    __syncthreads();

    if (blockDim.x >= 512 && tid < 256) idata[tid] += idata[tid + 256];
    __syncthreads();

    if (blockDim.x >= 256 && tid < 128) idata[tid] += idata[tid + 128];
    __syncthreads();

    if (blockDim.x >= 128 && tid < 64) idata[tid] += idata[tid + 64];
    __syncthreads();

    // unrolling warp
    if (tid < 32) {
        volatile float *vmem = idata;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid +  8];
        vmem[tid] += vmem[tid +  4];
        vmem[tid] += vmem[tid +  2];
        vmem[tid] += vmem[tid +  1];
    }

    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

template <int iBlockSize>
__global__ void reduceCompleteUnroll(float *g_idata, float * g_odata, const int n) {
    const int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;
    if (idx >= n) return;
    const int tid = threadIdx.x;

    // convert global data pointer to the local pointer of this block
    float *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // unrolling 8
    if (idx + 7 * blockDim.x < n) {
        float a1 = g_idata[idx];
        float a2 = g_idata[idx + blockDim.x];
        float a3 = g_idata[idx + 2 * blockDim.x];
        float a4 = g_idata[idx + 3 * blockDim.x];
        float a5 = g_idata[idx + 4 * blockDim.x];
        float a6 = g_idata[idx + 5 * blockDim.x];
        float a7 = g_idata[idx + 6 * blockDim.x];
        float a8 = g_idata[idx + 7 * blockDim.x];
        g_idata[idx] = a1 + a2 + a3 + a4 + a5 + a6 + a7 + a8;
    }
    __syncthreads();

    // in-place reduction and complete unroll
    if (iBlockSize >= 1024 && tid < 512) idata[tid] += idata[tid + 512];
    __syncthreads();

    if (iBlockSize >= 512 && tid < 256) idata[tid] += idata[tid + 256];
    __syncthreads();

    if (iBlockSize >= 256 && tid < 128) idata[tid] += idata[tid + 128];
    __syncthreads();

    if (iBlockSize >= 128 && tid < 64) idata[tid] += idata[tid + 64];
    __syncthreads();

    // unrolling warp
    if (tid < 32) {
        volatile float *vmem = idata;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid +  8];
        vmem[tid] += vmem[tid +  4];
        vmem[tid] += vmem[tid +  2];
        vmem[tid] += vmem[tid +  1];
    }

    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

/**********host functions**********/

void initialData(float *ip, const int size) {
    for (int i = 0; i < size; i++) {
        ip[i] = (float) (rand() & 0xFF);
    }
}

float naiveReduce(float *data, int size) {
    float sum = 0.0f;
    for (int i = 0; i < size; i++) {
        sum += data[i];
    }
    return sum;
}

float neighboredPairReduce(float *data, const int size) {
    for (int stride = 1; stride <= size / 2; stride *= 2) {
        for (int i = 0; i < size; i += stride * 2) {
            data[i] += data[i + stride];
        }
    }
    return data[0];
}

float interleavedPairReduce(float *data, const int size) {
    if (size == 1) return data[0];
    const int stride = size / 2;
    for (int i = 0; i < stride; i++) {
        data[i] += data[i + stride];
    }
    return interleavedPairReduce(data, stride);
}