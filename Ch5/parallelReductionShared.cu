#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define CHECK(call) {                                                        \
    const hipError_t error = call;                                          \
    if (error != hipSuccess) {                                              \
        printf("Error: %s:%d, ", __FILE__, __LINE__);                        \
        printf("code: %d, reason: %s\n", error, hipGetErrorString(error));  \
        exit(1);                                                             \
    }                                                                        \
}                                                                            \

void initialData(int *ip, const int size);
int naiveReduce(int *data, int size);
int neighboredPairReduce(int *data, const int size);
int interleavedPairReduce(int *data, const int size);

__global__ void reduceCompleteUnrollWarps8Sync(int *g_idata, int *g_odata, const int n);

int main(int argc, char **argv) {
    int size = 1<<24, evenSize = size;
    if (evenSize % 2 != 0) evenSize++; // should be even for pair-reducution to work
    printf("Vector size %d\n", size);

    size_t nBytes = evenSize * sizeof(int);
    clock_t start, end;
    double exeTime;
    int reductionSum;

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp; 
    CHECK(hipGetDeviceProperties(&deviceProp, dev)); 
    printf("Using Device %d: %s\n", dev, deviceProp.name); 
    CHECK(hipSetDevice(dev));

    // grid and block configuration
    int blockSize = 512;
    if (argc > 1) blockSize = atoi(argv[1]);
    dim3 block(blockSize);
    dim3 grid((evenSize + block.x - 1)/ block.x);
    printf("Grid dimension %d Block dimensiton %d\n", grid.x, block.x);

    // allocate host memory
    int *h_idata, *h_odata, *h_idata_cpy;
    h_idata = (int *) malloc(nBytes);
    h_odata = (int *) malloc(grid.x * sizeof(int));
    h_idata_cpy = (int *) malloc(nBytes);
    memset(h_idata, 0, nBytes);
    initialData(h_idata, size);
    memcpy(h_idata_cpy, h_idata, nBytes);

    // 0. compute on CPU
    start = clock();
    // reductionSum = naiveReduce(h_idata_cpy, size);
    // reductionSum = neighboredPairReduce(h_idata_cpy, evenSize);
    reductionSum = interleavedPairReduce(h_idata_cpy, evenSize);
    end = clock();
    exeTime = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("\nCPU reduce:                   execution time %.4f ms, result %d\n\n", exeTime * 1e3, reductionSum);

    // allocate device memory
    int *d_idata, *d_odata;
    CHECK(hipMalloc((int**)&d_idata, nBytes));
    CHECK(hipMalloc((int**)&d_odata, grid.x * sizeof(int)));

    // 1. baseline - not using shared memory 
    CHECK(hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice));
    start = clock();
    // CUDA part
    reduceCompleteUnrollWarps8<<<grid.x / 8, block>>>(d_idata, d_odata, evenSize);
    CHECK(hipDeviceSynchronize());
    end = clock();
    exeTime = ((double) (end - start)) / CLOCKS_PER_SEC;
    // Host part
    reductionSum = 0;
    CHECK(hipMemcpy(h_odata, d_odata, grid.x / 8 * sizeof(int), hipMemcpyDeviceToHost));
    for (int i = 0; i < grid.x / 8; i++) reductionSum += h_odata[i];
    printf("GPU complete unrolling:       execution time %.4f ms, result %d\n", exeTime * 1e3, reductionSum);

    // free host mem
    free(h_idata);
    free(h_odata);
    free(h_idata_cpy);

    // free device mem
    CHECK(hipFree(d_idata));
    CHECK(hipFree(d_odata));
    
    // clean up all resources
    CHECK(hipDeviceReset());
    return 0;
}

/**********CUDA kernels**********/

// baseline; not using shared memory
__global__ void reduceCompleteUnrollWarps8(int *g_idata, int * g_odata, const int n) {
    const int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;
    if (idx >= n) return;
    const int tid = threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // unrolling 8
    if (idx + 7 * blockDim.x < n) {
        int a1 = g_idata[idx];
        int a2 = g_idata[idx + blockDim.x];
        int a3 = g_idata[idx + 2 * blockDim.x];
        int a4 = g_idata[idx + 3 * blockDim.x];
        int a5 = g_idata[idx + 4 * blockDim.x];
        int a6 = g_idata[idx + 5 * blockDim.x];
        int a7 = g_idata[idx + 6 * blockDim.x];
        int a8 = g_idata[idx + 7 * blockDim.x];
        g_idata[idx] = a1 + a2 + a3 + a4 + a5 + a6 + a7 + a8;
    }
    __syncthreads();

    // in-place reduction and complete unroll
    if (blockDim.x >= 1024 && tid < 512) idata[tid] += idata[tid + 512];
    __syncthreads();

    if (blockDim.x >= 512 && tid < 256) idata[tid] += idata[tid + 256];
    __syncthreads();

    if (blockDim.x >= 256 && tid < 128) idata[tid] += idata[tid + 128];
    __syncthreads();

    if (blockDim.x >= 128 && tid < 64) idata[tid] += idata[tid + 64];
    __syncthreads();

    // unrolling wrap
    if (tid < 32) {
        volatile int *vmem = idata;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid +  8];
        vmem[tid] += vmem[tid +  4];
        vmem[tid] += vmem[tid +  2];
        vmem[tid] += vmem[tid +  1];
    }

    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

// using shared memory

/**********host functions**********/

void initialData(int *ip, const int size) {
    // generate different seed for random number
    time_t t;
    srand((unsigned int) time(&t));

    for (int i = 0; i < size; i++) {
        ip[i] = (int)( rand() & 0xFF );
    }
}

int naiveReduce(int *data, int size) {
    int sum = 0;
    for (int i = 0; i < size; i++) {
        sum += data[i];
    }
    return sum;
}

int neighboredPairReduce(int *data, const int size) {
    for (int stride = 1; stride <= size / 2; stride *= 2) {
        for (int i = 0; i < size; i += stride * 2) {
            data[i] += data[i + stride];
        }
    }
    return data[0];
}

int interleavedPairReduce(int *data, const int size) {
    if (size == 1) return data[0];
    const int stride = size / 2;
    for (int i = 0; i < stride; i++) {
        data[i] += data[i + stride];
    }
    return interleavedPairReduce(data, stride);
}