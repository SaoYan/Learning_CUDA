#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define CHECK(call) {                                                        \
    const hipError_t error = call;                                          \
    if (error != hipSuccess) {                                              \
        printf("Error: %s:%d, ", __FILE__, __LINE__);                        \
        printf("code: %d, reason: %s\n", error, hipGetErrorString(error));  \
        exit(1);                                                             \
    }                                                                        \
}                                                                            \

#define BLOCKSIZE 256

void initialData(int *ip, const int size);
int naiveReduce(int *data, int size);
int neighboredPairReduce(int *data, const int size);
int interleavedPairReduce(int *data, const int size);

__global__ void warmup(int *g_idata, int * g_odata, const int n);

__global__ void reduceGlobalMem(int *g_idata, int * g_odata, const int n);
__global__ void reduceGlobalMemUnroll8(int *g_idata, int * g_odata, const int n);

__global__ void reduceSharedMem(int *g_idata, int * g_odata, const int n);
__global__ void reduceSharedMemUnroll8(int *g_idata, int * g_odata, const int n);
__global__ void reduceSharedMemDynUnroll8(int *g_idata, int * g_odata, const int n);

int main(int argc, char **argv) {
    int size = 1<<24, evenSize = size;
    if (evenSize % 2 != 0) evenSize++; // should be even for pair-reducution to work
    printf("Vector size %d\n", size);

    size_t nBytes = evenSize * sizeof(int);
    clock_t start, end;
    double exeTime;
    int reductionSum;

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp; 
    CHECK(hipGetDeviceProperties(&deviceProp, dev)); 
    printf("Using Device %d: %s\n", dev, deviceProp.name); 
    CHECK(hipSetDevice(dev));

    // grid and block configuration
    dim3 block(BLOCKSIZE);
    dim3 grid((evenSize + block.x - 1)/ block.x);
    printf("Grid dimension %d Block dimensiton %d\n", grid.x, block.x);

    // allocate host memory
    int *h_idata, *h_odata, *h_idata_cpy;
    h_idata = (int *) malloc(nBytes);
    h_odata = (int *) malloc(grid.x * sizeof(int));
    h_idata_cpy = (int *) malloc(nBytes);
    memset(h_idata, 0, nBytes);
    initialData(h_idata, size);
    memcpy(h_idata_cpy, h_idata, nBytes);

    // 0. compute on CPU
    start = clock();
    // reductionSum = naiveReduce(h_idata_cpy, size);
    // reductionSum = neighboredPairReduce(h_idata_cpy, evenSize);
    reductionSum = interleavedPairReduce(h_idata_cpy, evenSize);
    end = clock();
    exeTime = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("\nCPU reduce: execution time %.4f ms, result %d\n\n", exeTime * 1e3, reductionSum);

    // allocate device memory
    int *d_idata, *d_odata;
    CHECK(hipMalloc((int**)&d_idata, nBytes));
    CHECK(hipMalloc((int**)&d_odata, grid.x * sizeof(int)));

    // warmup
    warmup<<<grid.x, block>>>(d_idata, d_odata, evenSize);
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());

    // 1.1 baseline - not using shared memory 
    CHECK(hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemset(d_odata, 0, grid.x * sizeof(int)));
    memset(h_odata, 0, grid.x * sizeof(int));
    start = clock();
    // CUDA part
    reduceGlobalMem<<<grid.x, block>>>(d_idata, d_odata, evenSize);
    CHECK(hipDeviceSynchronize());
    end = clock();
    exeTime = ((double) (end - start)) / CLOCKS_PER_SEC;
    // Host part
    reductionSum = 0;
    CHECK(hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost));
    for (int i = 0; i < grid.x; i++) reductionSum += h_odata[i];
    printf("GPU baseline:      execution time %.4f ms, result %d\n", exeTime * 1e3, reductionSum);
    CHECK(hipGetLastError());

    // using shared memory 
    CHECK(hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemset(d_odata, 0, grid.x * sizeof(int)));
    memset(h_odata, 0, grid.x * sizeof(int));
    start = clock();
    // CUDA part
    reduceSharedMem<<<grid.x, block>>>(d_idata, d_odata, evenSize);
    CHECK(hipDeviceSynchronize());
    end = clock();
    exeTime = ((double) (end - start)) / CLOCKS_PER_SEC;
    // Host part
    reductionSum = 0;
    CHECK(hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost));
    for (int i = 0; i < grid.x; i++) reductionSum += h_odata[i];
    printf("GPU shared memory: execution time %.4f ms, result %d\n\n", exeTime * 1e3, reductionSum);
    CHECK(hipGetLastError());

    // baseline - not using shared memory + x8 unrolling 
    CHECK(hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemset(d_odata, 0, grid.x * sizeof(int)));
    memset(h_odata, 0, grid.x * sizeof(int));
    start = clock();
    // CUDA part
    reduceGlobalMemUnroll8<<<grid.x / 8, block>>>(d_idata, d_odata, evenSize);
    CHECK(hipDeviceSynchronize());
    end = clock();
    exeTime = ((double) (end - start)) / CLOCKS_PER_SEC;
    // Host part
    reductionSum = 0;
    CHECK(hipMemcpy(h_odata, d_odata, grid.x / 8 * sizeof(int), hipMemcpyDeviceToHost));
    for (int i = 0; i < grid.x / 8; i++) reductionSum += h_odata[i];
    printf("GPU baseline x8 unrolling:              execution time %.4f ms, result %d\n", exeTime * 1e3, reductionSum);
    CHECK(hipGetLastError());

    // using shared memory + x8 unrolling  
    CHECK(hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemset(d_odata, 0, grid.x * sizeof(int)));
    memset(h_odata, 0, grid.x * sizeof(int));
    start = clock();
    // CUDA part
    reduceSharedMemUnroll8<<<grid.x / 8, block>>>(d_idata, d_odata, evenSize);
    CHECK(hipDeviceSynchronize());
    end = clock();
    exeTime = ((double) (end - start)) / CLOCKS_PER_SEC;
    // Host part
    reductionSum = 0;
    CHECK(hipMemcpy(h_odata, d_odata, grid.x / 8 * sizeof(int), hipMemcpyDeviceToHost));
    for (int i = 0; i < grid.x / 8; i++) reductionSum += h_odata[i];
    printf("GPU shared memory x8 unrolling:         execution time %.4f ms, result %d\n", exeTime * 1e3, reductionSum);
    CHECK(hipGetLastError());

    // GPU dynamic shared memory + x8 unrolling  
    CHECK(hipMemcpy(d_idata, h_idata, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemset(d_odata, 0, grid.x * sizeof(int)));
    memset(h_odata, 0, grid.x * sizeof(int));
    start = clock();
    // CUDA part
    reduceSharedMemDynUnroll8<<<grid.x / 8, block, BLOCKSIZE * sizeof(int)>>>(d_idata, d_odata, evenSize);
    CHECK(hipDeviceSynchronize());
    end = clock();
    exeTime = ((double) (end - start)) / CLOCKS_PER_SEC;
    // Host part
    reductionSum = 0;
    CHECK(hipMemcpy(h_odata, d_odata, grid.x / 8 * sizeof(int), hipMemcpyDeviceToHost));
    for (int i = 0; i < grid.x / 8; i++) reductionSum += h_odata[i];
    printf("GPU dynamic shared memory x8 unrolling: execution time %.4f ms, result %d\n", exeTime * 1e3, reductionSum);
    CHECK(hipGetLastError());

    // free host mem
    free(h_idata);
    free(h_odata);
    free(h_idata_cpy);

    // free device mem
    CHECK(hipFree(d_idata));
    CHECK(hipFree(d_odata));
    
    // clean up all resources
    CHECK(hipDeviceReset());
    return 0;
}

/**********CUDA kernels**********/

__global__ void warmup(int *g_idata, int * g_odata, const int n) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    const int tid = threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // in-place reduction and complete unroll
    if (blockDim.x >= 1024 && tid < 512) idata[tid] += idata[tid + 512];
    __syncthreads();

    if (blockDim.x >= 512 && tid < 256) idata[tid] += idata[tid + 256];
    __syncthreads();

    if (blockDim.x >= 256 && tid < 128) idata[tid] += idata[tid + 128];
    __syncthreads();

    if (blockDim.x >= 128 && tid < 64) idata[tid] += idata[tid + 64];
    __syncthreads();

    // unrolling warp
    if (tid < 32) {
        volatile int *vmem = idata;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid +  8];
        vmem[tid] += vmem[tid +  4];
        vmem[tid] += vmem[tid +  2];
        vmem[tid] += vmem[tid +  1];
    }

    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

// baseline - not using shared memory
__global__ void reduceGlobalMem(int *g_idata, int * g_odata, const int n) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    const int tid = threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // in-place reduction and complete unroll
    if (blockDim.x >= 1024 && tid < 512) idata[tid] += idata[tid + 512];
    __syncthreads();

    if (blockDim.x >= 512 && tid < 256) idata[tid] += idata[tid + 256];
    __syncthreads();

    if (blockDim.x >= 256 && tid < 128) idata[tid] += idata[tid + 128];
    __syncthreads();

    if (blockDim.x >= 128 && tid < 64) idata[tid] += idata[tid + 64];
    __syncthreads();

    // unrolling warp
    if (tid < 32) {
        volatile int *vmem = idata;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid +  8];
        vmem[tid] += vmem[tid +  4];
        vmem[tid] += vmem[tid +  2];
        vmem[tid] += vmem[tid +  1];
    }

    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceGlobalMemUnroll8(int *g_idata, int * g_odata, const int n) {
    const int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;
    if (idx >= n) return;
    const int tid = threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // unrolling 8
    if (idx + 7 * blockDim.x < n) {
        int a1 = g_idata[idx];
        int a2 = g_idata[idx + blockDim.x];
        int a3 = g_idata[idx + 2 * blockDim.x];
        int a4 = g_idata[idx + 3 * blockDim.x];
        int a5 = g_idata[idx + 4 * blockDim.x];
        int a6 = g_idata[idx + 5 * blockDim.x];
        int a7 = g_idata[idx + 6 * blockDim.x];
        int a8 = g_idata[idx + 7 * blockDim.x];
        g_idata[idx] = a1 + a2 + a3 + a4 + a5 + a6 + a7 + a8;
    }
    __syncthreads();

    // in-place reduction and complete unroll
    if (blockDim.x >= 1024 && tid < 512) idata[tid] += idata[tid + 512];
    __syncthreads();

    if (blockDim.x >= 512 && tid < 256) idata[tid] += idata[tid + 256];
    __syncthreads();

    if (blockDim.x >= 256 && tid < 128) idata[tid] += idata[tid + 128];
    __syncthreads();

    if (blockDim.x >= 128 && tid < 64) idata[tid] += idata[tid + 64];
    __syncthreads();

    // unrolling warp
    if (tid < 32) {
        volatile int *vmem = idata;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid +  8];
        vmem[tid] += vmem[tid +  4];
        vmem[tid] += vmem[tid +  2];
        vmem[tid] += vmem[tid +  1];
    }

    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

// using shared memory
__global__ void reduceSharedMem(int *g_idata, int * g_odata, const int n) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    const int tid = threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // shared memory
    __shared__ int smem[BLOCKSIZE];
    smem[tid] = idata[tid];
    __syncthreads();

    // in-place reduction and complete unroll
    if (blockDim.x >= 1024 && tid < 512) smem[tid] += smem[tid + 512];
    __syncthreads();

    if (blockDim.x >= 512 && tid < 256) smem[tid] += smem[tid + 256];
    __syncthreads();

    if (blockDim.x >= 256 && tid < 128) smem[tid] += smem[tid + 128];
    __syncthreads();

    if (blockDim.x >= 128 && tid < 64) smem[tid] += smem[tid + 64];
    __syncthreads();

    // unrolling warp
    if (tid < 32) {
        volatile int *vmem = smem;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid +  8];
        vmem[tid] += vmem[tid +  4];
        vmem[tid] += vmem[tid +  2];
        vmem[tid] += vmem[tid +  1];
    }

    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}

__global__ void reduceSharedMemUnroll8(int *g_idata, int * g_odata, const int n) {
    const int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;
    if (idx >= n) return;
    const int tid = threadIdx.x;

    // unrolling 8
    __shared__ int smem[BLOCKSIZE];
    int temp = 0;
    if (idx + 7 * blockDim.x < n) {
        int a1 = g_idata[idx];
        int a2 = g_idata[idx + blockDim.x];
        int a3 = g_idata[idx + 2 * blockDim.x];
        int a4 = g_idata[idx + 3 * blockDim.x];
        int a5 = g_idata[idx + 4 * blockDim.x];
        int a6 = g_idata[idx + 5 * blockDim.x];
        int a7 = g_idata[idx + 6 * blockDim.x];
        int a8 = g_idata[idx + 7 * blockDim.x];
        temp = a1 + a2 + a3 + a4 + a5 + a6 + a7 + a8;
    }
    smem[tid] = temp;
    __syncthreads();

    // in-place reduction and complete unroll
    if (blockDim.x >= 1024 && tid < 512) smem[tid] += smem[tid + 512];
    __syncthreads();

    if (blockDim.x >= 512 && tid < 256) smem[tid] += smem[tid + 256];
    __syncthreads();

    if (blockDim.x >= 256 && tid < 128) smem[tid] += smem[tid + 128];
    __syncthreads();

    if (blockDim.x >= 128 && tid < 64) smem[tid] += smem[tid + 64];
    __syncthreads();

    // unrolling warp
    if (tid < 32) {
        volatile int *vmem = smem;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid +  8];
        vmem[tid] += vmem[tid +  4];
        vmem[tid] += vmem[tid +  2];
        vmem[tid] += vmem[tid +  1];
    }

    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}

__global__ void reduceSharedMemDynUnroll8(int *g_idata, int * g_odata, const int n) {
    const int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;
    if (idx >= n) return;
    const int tid = threadIdx.x;

    // unrolling 8
    __shared__ int smem[BLOCKSIZE];
    int temp = 0;
    if (idx + 7 * blockDim.x < n) {
        int a1 = g_idata[idx];
        int a2 = g_idata[idx + blockDim.x];
        int a3 = g_idata[idx + 2 * blockDim.x];
        int a4 = g_idata[idx + 3 * blockDim.x];
        int a5 = g_idata[idx + 4 * blockDim.x];
        int a6 = g_idata[idx + 5 * blockDim.x];
        int a7 = g_idata[idx + 6 * blockDim.x];
        int a8 = g_idata[idx + 7 * blockDim.x];
        temp = a1 + a2 + a3 + a4 + a5 + a6 + a7 + a8;
    }
    smem[tid] = temp;
    __syncthreads();

    // in-place reduction and complete unroll
    if (blockDim.x >= 1024 && tid < 512) smem[tid] += smem[tid + 512];
    __syncthreads();

    if (blockDim.x >= 512 && tid < 256) smem[tid] += smem[tid + 256];
    __syncthreads();

    if (blockDim.x >= 256 && tid < 128) smem[tid] += smem[tid + 128];
    __syncthreads();

    if (blockDim.x >= 128 && tid < 64) smem[tid] += smem[tid + 64];
    __syncthreads();

    // unrolling warp
    if (tid < 32) {
        volatile int *vmem = smem;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid +  8];
        vmem[tid] += vmem[tid +  4];
        vmem[tid] += vmem[tid +  2];
        vmem[tid] += vmem[tid +  1];
    }

    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}

/**********host functions**********/

void initialData(int *ip, const int size) {
    // generate different seed for random number
    time_t t;
    srand((unsigned int) time(&t));

    for (int i = 0; i < size; i++) {
        ip[i] = (int)( rand() & 0xFF );
    }
}

int naiveReduce(int *data, int size) {
    int sum = 0;
    for (int i = 0; i < size; i++) {
        sum += data[i];
    }
    return sum;
}

int neighboredPairReduce(int *data, const int size) {
    for (int stride = 1; stride <= size / 2; stride *= 2) {
        for (int i = 0; i < size; i += stride * 2) {
            data[i] += data[i + stride];
        }
    }
    return data[0];
}

int interleavedPairReduce(int *data, const int size) {
    if (size == 1) return data[0];
    const int stride = size / 2;
    for (int i = 0; i < stride; i++) {
        data[i] += data[i + stride];
    }
    return interleavedPairReduce(data, stride);
}