#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define CHECK(call) {                                                        \
    const hipError_t error = call;                                          \
    if (error != hipSuccess) {                                              \
        printf("Error: %s:%d, ", __FILE__, __LINE__);                        \
        printf("code: %d, reason: %s\n", error, hipGetErrorString(error));  \
        exit(1);                                                             \
    }                                                                        \
}                                                                            \

#define BDIMX 16
#define SEGM  4

void printData(int *data, const int size);



int main(int argc, char **argv) {
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp; 
    CHECK(hipGetDeviceProperties(&deviceProp, dev)); 
    printf("Using Device %d: %s\n", dev, deviceProp.name); 
    CHECK(hipSetDevice(dev));

    int nElem = BDIMX;
    int h_in[BDIMX], h_out[BDIMX];
    for (int i = 0; i < nElem; i++) h_in[i] = i;
    printf("initialData\t\t: ");
    printData(h_in, nElem);

    size_t nBytes = nElem * sizeof(int);
    int *d_in, *d_out;
    CHECK(hipMalloc((int**)&d_in, nBytes));
    CHECK(hipMalloc((int**)&d_out, nBytes));
    CHECK(hipMemcpy(d_in, h_in, nBytes, hipMemcpyHostToDevice));
    dim3 block(BDIMX);

    // Broadcasting a value across a warp
    hipMemset(d_out, 0, nBytes);
    memset(h_out, 0, nBytes);
    shfl_broadcast<<<1, block>>>(d_out, d_in, 2);
    CHECK(hipGetLastError());
    CHECK(hipMemcpy(h_out, d_out, nBytes, hipMemcpyDeviceToHost));
    printf("broadcast\t\t: ");
    printData(h_out, nElem);

    CHECK(hipFree(d_in));
    CHECK(hipFree(d_out));
    CHECK(hipDeviceReset());
    return 0;
}

/**********host functions**********/

void printData(int *data, const int size) {
    for (int i = 0; i < size; i++) {
        printf("%d ", data[i]);
    }
    printf("\n");
}

/**********CUDA kernels**********/

__global__ void shfl_broadcast(int *out, int*in, int srcLane) {
    int value = in[threadIdx.x];
    value = __shfl(value, srcLane, blockDim.x);
    out[threadIdx.x] = value;
}
