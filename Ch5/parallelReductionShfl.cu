#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define CHECK(call) {                                                        \
    const hipError_t error = call;                                          \
    if (error != hipSuccess) {                                              \
        printf("Error: %s:%d, ", __FILE__, __LINE__);                        \
        printf("code: %d, reason: %s\n", error, hipGetErrorString(error));  \
        exit(1);                                                             \
    }                                                                        \
}                                                                            \

#define BLOCKSIZE 256

void initialData(int *ip, const int size);
int naiveReduce(int *data, int size);
int neighboredPairReduce(int *data, const int size);
int interleavedPairReduce(int *data, const int size);

int main(int argc, char **argv) {
    int size = 1<<24, evenSize = size;
    if (evenSize % 2 != 0) evenSize++; // should be even for pair-reducution to work
    printf("Vector size %d\n", size);

    size_t nBytes = evenSize * sizeof(int);
    clock_t start, end;
    double exeTime;
    int reductionSum;

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp; 
    CHECK(hipGetDeviceProperties(&deviceProp, dev)); 
    printf("Using Device %d: %s\n", dev, deviceProp.name); 
    CHECK(hipSetDevice(dev));

    // grid and block configuration
    dim3 block(BLOCKSIZE);
    dim3 grid((evenSize + block.x - 1)/ block.x);
    printf("Grid dimension %d Block dimensiton %d\n", grid.x, block.x);

    // allocate host memory
    int *h_idata, *h_odata, *h_idata_cpy;
    h_idata = (int *) malloc(nBytes);
    h_odata = (int *) malloc(grid.x * sizeof(int));
    h_idata_cpy = (int *) malloc(nBytes);
    memset(h_idata, 0, nBytes);
    initialData(h_idata, size);
    memcpy(h_idata_cpy, h_idata, nBytes);

    // 0. compute on CPU
    start = clock();
    // reductionSum = naiveReduce(h_idata_cpy, size);
    // reductionSum = neighboredPairReduce(h_idata_cpy, evenSize);
    reductionSum = interleavedPairReduce(h_idata_cpy, evenSize);
    end = clock();
    exeTime = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("\nCPU reduce: execution time %.4f ms, result %d\n\n", exeTime * 1e3, reductionSum);

    // allocate device memory
    int *d_idata, *d_odata;
    CHECK(hipMalloc((int**)&d_idata, nBytes));
    CHECK(hipMalloc((int**)&d_odata, grid.x * sizeof(int)));

    // free host mem
    free(h_idata);
    free(h_odata);
    free(h_idata_cpy);

    // free device mem
    CHECK(hipFree(d_idata));
    CHECK(hipFree(d_odata));
    
    // clean up all resources
    CHECK(hipDeviceReset());
    return 0;
}


/**********host functions**********/

void initialData(int *ip, const int size) {
    // generate different seed for random number
    time_t t;
    srand((unsigned int) time(&t));

    for (int i = 0; i < size; i++) {
        ip[i] = (int)( rand() & 0xFF );
    }
}

int naiveReduce(int *data, int size) {
    int sum = 0;
    for (int i = 0; i < size; i++) {
        sum += data[i];
    }
    return sum;
}

int neighboredPairReduce(int *data, const int size) {
    for (int stride = 1; stride <= size / 2; stride *= 2) {
        for (int i = 0; i < size; i += stride * 2) {
            data[i] += data[i + stride];
        }
    }
    return data[0];
}

int interleavedPairReduce(int *data, const int size) {
    if (size == 1) return data[0];
    const int stride = size / 2;
    for (int i = 0; i < stride; i++) {
        data[i] += data[i + stride];
    }
    return interleavedPairReduce(data, stride);
}