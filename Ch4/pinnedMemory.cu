#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define CHECK(call) {                                                        \
    const hipError_t error = call;                                          \
    if (error != hipSuccess) {                                              \
        printf("Error: %s:%d, ", __FILE__, __LINE__);                        \
        printf("code: %d, reason: %s\n", error, hipGetErrorString(error));  \
        exit(1);                                                             \
    }                                                                        \
}                                                                            \

int main(int argc, char **argv) {
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp; 
    CHECK(hipGetDeviceProperties(&deviceProp, dev)); 
    printf("Using Device %d: %s\n", dev, deviceProp.name); 
    CHECK(hipSetDevice(dev));

    if (!deviceProp.canMapHostMemory) {
        printf("Device %d does not support mapping CPU host memory!\n", dev);
        CHECK(hipDeviceReset());
        exit(EXIT_SUCCESS);
    }

    int size = 1 << 30;
    size_t nBytes = size * sizeof(float);
    clock_t start, end;
    double time;

    // allocate host memory
    float *h_a, *h_a_pin;
    h_a = (float *) malloc(nBytes);
    CHECK(hipHostMalloc((float **)&h_a_pin, nBytes)); // pinned memory
    memset(h_a, 0, nBytes);
    memset(h_a_pin, 0, nBytes);

    // allocate device memory
    float *d_a1, *d_a2;
    CHECK(hipMalloc((float **)&d_a1, nBytes));
    CHECK(hipMalloc((float **)&d_a2, nBytes));

    // pageable memory <--> device
    start = clock();
    CHECK(hipMemcpy(d_a1, h_a, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(h_a, d_a1, nBytes, hipMemcpyDeviceToHost));
    end = clock();
    time = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("data transfer between pageable memory and device takes %.4f ms\n", time * 1000);

    // pinned memory <--> device
    start = clock();
    CHECK(hipMemcpy(d_a2, h_a_pin, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(h_a_pin, d_a2, nBytes, hipMemcpyDeviceToHost));
    end = clock();
    time = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("data transfer between pinned memory and device takes %.4f ms\n", time * 1000);

    free(h_a);
    CHECK(hipHostFree(h_a_pin));
    CHECK(hipFree(d_a1));
    CHECK(hipFree(d_a2));

    CHECK(hipDeviceReset());
    return 0;
}