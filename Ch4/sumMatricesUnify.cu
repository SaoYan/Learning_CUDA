#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

void initialData(float *ip, const int nx, const int ny);
void sumArraysOnHost(float *A, float *B, float *C, const int nx, const int ny);
void verifyResult(float *hostRes, float *deviceRes, const int nx, const int ny);

__global__ void sumArraysOnDevice(float *A, float *B, float *C, const int nx, const int ny);

#define CHECK(call) {                                                        \
    const hipError_t error = call;                                          \
    if (error != hipSuccess) {                                              \
        printf("Error: %s:%d, ", __FILE__, __LINE__);                        \
        printf("code: %d, reason: %s\n", error, hipGetErrorString(error));  \
        exit(1);                                                             \
    }                                                                        \
}                                                                            \

int main(int argc, char **argv) {
    int nx = 1<<14, ny = 1<<14;
    size_t nBytes = nx * ny * sizeof(float);
    clock_t start, end;
    printf("Matrix size (%d, %d)\n", nx, ny);

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp; 
    CHECK(hipGetDeviceProperties(&deviceProp, dev)); 
    printf("Using Device %d: %s\n", dev, deviceProp.name); 
    CHECK(hipSetDevice(dev));

    // allocate unified memory
    float *A, *B, *C, *C_gpu;
    CHECK(hipMallocManaged((float**)&A, nBytes));
    CHECK(hipMallocManaged((float**)&B, nBytes));
    CHECK(hipMallocManaged((float**)&C, nBytes));
    CHECK(hipMallocManaged((float**)&C_gpu, nBytes));

    // initialize data
    initialData(A, nx, ny);
    initialData(B, nx, ny);
    memset(C, 0, nBytes);
    memset(C_gpu, 0, nBytes);

    // compute on CPU
    start = clock();
    sumArraysOnHost(A, B, C, nx, ny);
    end = clock();
    double cpuTime = ((double) (end - start)) / CLOCKS_PER_SEC;

    // configration
    int dimx = 16, dimy = 16;
    dim3 block(dimx, dimy);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
    printf("Grid dimension (%d, %d) Block dimensiton (%d, %d)\n",grid.x, grid.y, block.x, block.y);

    // launch CUDA kernel
    memset(C_gpu, 0, nBytes);
    start = clock();
    sumArraysOnDevice<<<grid, block>>>(A, B, C_gpu, nx, ny);
    CHECK(hipDeviceSynchronize()); // synchronization is necessary when using unified memory!
    end = clock();
    double gpuTime = ((double) (end - start)) / CLOCKS_PER_SEC;

    // verify
    verifyResult(C, C_gpu, nx, ny);
    CHECK(hipGetLastError());
    printf("It takes %.4f sec to execute on CPU\n", cpuTime);
    printf("It takes %.4f sec to execute on GPU\n", gpuTime);

    // free memory
    CHECK(hipFree(A));
    CHECK(hipFree(B));
    CHECK(hipFree(C));
    CHECK(hipFree(C_gpu));
    
    // clean up all resources
    CHECK(hipDeviceReset());
    return 0;
}

/**********CUDA kernels**********/

__global__ void sumArraysOnDevice(float *A, float *B, float *C, const int nx, const int ny) {
    // Thread and block index --> Coordinate in the matrix
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    // Coordinate in the matrix --> Offset in linear global memory  
    if (ix < nx && iy < ny)  {
        int idx = iy * nx + ix;
        C[idx] = A[idx] + B[idx];
    }
}

/**********host functions**********/

void initialData(float *ip, const int nx, const int ny) {
    // generate different seed for random number
    time_t t;
    srand((unsigned int) time(&t));

    for (int iy = 0; iy < ny; iy++) {
        for (int ix = 0; ix < nx; ix++) {
            int idx = iy * nx + ix;
            ip[idx] = (float) (rand() & 0xFF) / 10.f;
        }
    }
}

void sumArraysOnHost(float *A, float *B, float *C, const int nx, const int ny) {
    for (int iy = 0; iy < ny; iy++) {
        for (int ix = 0; ix < nx; ix++) {
            int idx = iy * nx + ix;
            C[idx] = A[idx] + B[idx];
        }
    }
}

void verifyResult(float *hostRes, float *deviceRes, const int nx, const int ny) {
    double eps = 1e-8;
    for (int iy = 0; iy < ny; iy++) {
        for (int ix = 0; ix < nx; ix++) {
            int idx = iy * nx + ix;
            if (abs(hostRes[idx] - deviceRes[idx]) > eps) {
                printf("Arrays do not match! Check your kernel code!\n");
                printf("host %5.2f gpu %5.2f at (%d, %d)\n", hostRes[idx], deviceRes[idx], nx, ny);
                return;
            }
        }
    }
    printf("Arrays match! Congrats, your kernel code works well!\n");
    return;
}