#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void checkGlobalVariable();

#define CHECK(call) {                                                        \
    const hipError_t error = call;                                          \
    if (error != hipSuccess) {                                              \
        printf("Error: %s:%d, ", __FILE__, __LINE__);                        \
        printf("code: %d, reason: %s\n", error, hipGetErrorString(error));  \
        exit(1);                                                             \
    }                                                                        \
}                                                                            \

__device__ float devData;

int main(int argc, char **argv) {
    // initialize the global variable
    float value = 3.14f;
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(devData), &value, sizeof(float)));
    printf("Host:   copied %f to the global variable\n", value);

    // invoke kernel
    checkGlobalVariable<<<1, 32>>>();
    
    // copy back to host
    CHECK(hipMemcpyFromSymbol(&value, HIP_SYMBOL(devData), sizeof(float)));
    printf("Host:   the value changed by the kernel to %f\n", value);

    hipDeviceReset();
    return 0;
}

__global__ void checkGlobalVariable() {
    printf("Device: the value of the global variable is %f\n", devData);
    devData += 2.0f;
    __syncthreads();
} 
