#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define CHECK(call) {                                                        \
    const hipError_t error = call;                                          \
    if (error != hipSuccess) {                                              \
        printf("Error: %s:%d, ", __FILE__, __LINE__);                        \
        printf("code: %d, reason: %s\n", error, hipGetErrorString(error));  \
        exit(1);                                                             \
    }                                                                        \
}                                                                            \

#define LEN 1<<22

typedef struct innerArray {
    float x[LEN];
    float y[LEN];
} innerArray;

void initialInnerArray(innerArray *ip,  int size);
void incrementInnerArrayOnHost(innerArray *input, innerArray *output, const int n);
void checkResult(innerArray *hostRef, innerArray *gpuRef, const int n);

__global__ void incrementInnerArray(innerArray *input, innerArray * output, const int n);

int main(int argc, char **argv) {
    int nElem = LEN;
    size_t nBytes = sizeof(innerArray);
    clock_t start, end;

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp; 
    CHECK(hipGetDeviceProperties(&deviceProp, dev)); 
    printf("Using Device %d: %s\n", dev, deviceProp.name); 
    CHECK(hipSetDevice(dev));

    // allocate host memory
    innerArray *h_in      = (innerArray *)malloc(nBytes);
    innerArray *h_out     = (innerArray *)malloc(nBytes);
    innerArray *h_out_gpu = (innerArray *)malloc(nBytes);
    initialInnerArray(h_in, nElem);

    // compute on CPU
    incrementInnerArrayOnHost(h_in, h_out, nElem);

    // allocate device memory
    innerArray *d_in, *d_out;
    CHECK(hipMalloc((innerArray**)&d_in, nBytes));
    CHECK(hipMalloc((innerArray**)&d_out, nBytes));

    // copy data from host to device
    CHECK(hipMemcpy(d_in, h_in, nBytes, hipMemcpyHostToDevice));

    // execute kernel
    int blocksize = 128;
    if (argc > 1) blocksize = atoi(argv[1]);
    dim3 block(blocksize, 1);
    dim3 grid((nElem + block.x - 1) / block.x, 1);
    start = clock();
    incrementInnerArray<<<grid, block>>>(d_in, d_out, nElem);
    CHECK(hipDeviceSynchronize()); // synchronize kernel only for debugging!
    end = clock();
    double time = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("<<< %3d, %3d >>> elapsed %f ms\n", grid.x, block.x, time * 1000.0);

    // copy data back to CPU
    CHECK(hipMemcpy(h_out_gpu, d_out, nBytes, hipMemcpyDeviceToHost));
    checkResult(h_out, h_out_gpu, nElem);
    CHECK(hipGetLastError());

    // free memories
    CHECK(hipFree(d_in));
    CHECK(hipFree(d_out));
    free(h_in);
    free(h_out);
    free(h_out_gpu);

    // reset device
    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}

/**********CUDA kernels**********/

__global__ void incrementInnerArray(innerArray *input, innerArray * output, const int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        float tmpx = input->x[i];
        float tmpy = input->y[i];
        output->x[i] = tmpx + 10.f;
        output->y[i] = tmpy + 20.f;
    }
}

/**********host functions**********/

void initialInnerArray(innerArray *ip,  int size) {
    for (int i = 0; i < size; i++) {
        ip->x[i] = (float)( rand() & 0xFF ) / 100.0f;
        ip->y[i] = (float)( rand() & 0xFF ) / 100.0f;
    }
    return;
}

void incrementInnerArrayOnHost(innerArray *input, innerArray *output, const int n)
{
    for (int i = 0; i < n; i++) {
        output->x[i] = input->x[i] + 10.f;
        output->y[i] = input->y[i] + 20.f;
    }
    return;
}

void checkResult(innerArray *hostRef, innerArray *gpuRef, const int n) {
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < n; i++) {
        if (abs(hostRef->x[i] - gpuRef->x[i]) > epsilon) {
            match = 0;
            printf("different on x %dth element: host %f gpu %f\n", 
                i, hostRef->x[i], gpuRef->x[i]);
            break;
        }

        if (abs(hostRef->y[i] - gpuRef->y[i]) > epsilon) {
            match = 0;
            printf("different on y %dth element: host %f gpu %f\n", 
                i, hostRef->y[i], gpuRef->y[i]);
            break;
        }
    }

    if (!match)  printf("Arrays do not match.\n\n");
}