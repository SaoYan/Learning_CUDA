#include <stdio.h>
#include <hip/hip_runtime.h>

void printValue(float *ip, const int n);
__global__ void modifyGlobalVariable(const int n);

#define CHECK(call) {                                                        \
    const hipError_t error = call;                                          \
    if (error != hipSuccess) {                                              \
        printf("Error: %s:%d, ", __FILE__, __LINE__);                        \
        printf("code: %d, reason: %s\n", error, hipGetErrorString(error));  \
        exit(1);                                                             \
    }                                                                        \
}                                                                            \

#define N 32

__device__ float devData[N];

int main(int argc, char **argv) {
    // initialize the global variable
    float *value = (float *)malloc(N * sizeof(float));
    for (int i = 0; i < N; i++) {
        value[i] = 3.14f;
    }
    printValue(value, N);
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(devData), value, N * sizeof(float)));

    // invoke kernel
    modifyGlobalVariable<<<1, N>>>(N);
    CHECK(hipDeviceSynchronize());
    
    // copy back to host
    CHECK(hipMemcpyFromSymbol(value, HIP_SYMBOL(devData), N * sizeof(float)));
    printValue(value, N);

    hipDeviceReset();
    return 0;
}

__global__ void modifyGlobalVariable(const int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        devData[idx] *= idx;
    }
} 

void printValue(float *ip, const int n) {
    for (int i = 0; i < n; i ++) {
        printf("%.2f, ", ip[i]);
    }
    printf("%c", '\n');
}
