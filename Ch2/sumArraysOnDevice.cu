#include <stdio.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>

void initialData(float *ip, int rows, int cols);
void sumArraysOnHost(float *A, float *B, float *C, const int rows, const int cols);
void verifyResult(float *hostRes, float *deviceRes, int rows, int cols);

__global__ void sumArraysOnDevice(float *A, float *B, float *C);

int main(int argc, char **argv) {
    // On current GPUs, a thread block may contain up to 1024 threads.
    int rows = 409600, cols = 512;
    size_t nBytes = rows * cols * sizeof(float);
    clock_t start, end;
    hipError_t error;

    // set up device 
    int dev = 0; 
    hipSetDevice(dev);

    // allocate host memory
    float *h_A, *h_B, *h_C, *h_C_gpu;
    h_A = (float *) malloc(nBytes);
    h_B = (float *) malloc(nBytes);
    h_C = (float *) malloc(nBytes);
    h_C_gpu = (float *) malloc(nBytes);

    // initial data (in CPU mem)
    initialData(h_A, rows, cols);
    initialData(h_B, rows, cols);
    memset(h_C, 0, nBytes);
    memset(h_C_gpu, 0, nBytes);

    // compute on CPU
    start = clock();
    sumArraysOnHost(h_A, h_B, h_C, rows, cols);
    end = clock();
    double cpuTime = ((double) (end - start)) / CLOCKS_PER_SEC;

    // allocate device mem
    float *d_A, *d_B, *d_C;
    hipMalloc((float**)&d_A, nBytes);
    hipMalloc((float**)&d_B, nBytes);
    error = hipMalloc((float**)&d_C, nBytes);
    printf("Allocating GPU mem: %s\n", hipGetErrorString(error));

    // copy data from CPU to GPU
    start = clock();
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    error = hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);
    end = clock();
    double copyTime = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("Copying data CPU --> GPU: %s\n", hipGetErrorString(error));

    // launch CUDA kernel
    dim3 block(cols);
    dim3 grid((rows*cols+block.x-1)/block.x);
    start = clock();
    sumArraysOnDevice<<<grid, block>>>(d_A, d_B, d_C);
    end = clock();
    double gpuTime = ((double) (end - start)) / CLOCKS_PER_SEC;
    
    // copy data from GPU back to CPU
    error = hipMemcpy(h_C_gpu, d_C, nBytes, hipMemcpyDeviceToHost);
    printf("Copyint data GPU --> CPU: %s\n", hipGetErrorString(error));

    // verify
    verifyResult(h_C, h_C_gpu, rows, cols);
    printf("It takes %.2f sec to execute on CPU\n", cpuTime);
    printf("It takes %.2f sec to copy data from CPU to GPU\n", copyTime);
    printf("It takes %.2f sec to execute on GPU\n", gpuTime);

    // free host mem
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C_gpu);

    // free device mem
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    // clean up all resources
    hipDeviceReset();

    return 0;
}

/**********CUDA kernels**********/

__global__ void sumArraysOnDevice(float *A, float *B, float *C) {
    // 1D grid of 1D block
    // compute global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    C[idx] = A[idx] + B[idx];
}

/**********host functions**********/

void initialData(float *ip, int rows, int cols) {
    // generate different seed for random number
    time_t t;
    srand((unsigned int) time(&t));

    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            int idx = i * cols + j;
            ip[idx] = (float) (rand() & 0xFF) / 10.f;
        }
    }
}

void sumArraysOnHost(float *A, float *B, float *C, const int rows, const int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            int idx = i * cols + j;
            C[idx] = A[idx] + B[idx];
        }
    }
}

void verifyResult(float *hostRes, float *deviceRes, int rows, int cols) {
    double eps = 1e-8;
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            int idx = i * cols + j;
            if (abs(hostRes[idx] - deviceRes[idx] > eps)) {
                printf("Arrays do not match! Check your kernel code!\n");
                printf("host %5.2f gpu %5.2f at (%d, %d)\n", hostRes[idx], deviceRes[idx], i, j);
                return;
            }
        }
    }
    printf("Arrays match! Congrats, your kernel code works well!\n");
    return;
}