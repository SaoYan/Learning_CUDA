#include <stdio.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>

void initialData(float *ip, int rows, int cols);
void sumArraysOnHost(float *A, float *B, float *C, const int rows, const int cols);
void verifyResult(float *hostRes, float *deviceRes, const int rows, const int cols);

__global__ void sumArraysOnDevice(float *A, float *B, float *C);

#define CHECK(call) {                                                        \
    const hipError_t error = call;                                          \
    if (error != hipSuccess) {                                              \
        printf("Error: %s:%d, ", __FILE__, __LINE__);                        \
        printf("code: %d, reason: %s\n", error, hipGetErrorString(error));  \
        exit(1);                                                             \
    }                                                                        \
}                                                                            \

int main(int argc, char **argv) {
    int rows = 409600, cols = 512;
    size_t nBytes = rows * cols * sizeof(float);
    clock_t start, end;

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp; 
    CHECK(hipGetDeviceProperties(&deviceProp, dev)); 
    printf("Using Device %d: %s\n", dev, deviceProp.name); 
    CHECK(hipSetDevice(dev));

    // allocate host memory
    float *h_A, *h_B, *h_C, *h_C_gpu;
    h_A = (float *) malloc(nBytes);
    h_B = (float *) malloc(nBytes);
    h_C = (float *) malloc(nBytes);
    h_C_gpu = (float *) malloc(nBytes);

    // initial data (in CPU mem)
    initialData(h_A, rows, cols);
    initialData(h_B, rows, cols);
    memset(h_C, 0, nBytes);
    memset(h_C_gpu, 0, nBytes);

    // compute on CPU
    start = clock();
    sumArraysOnHost(h_A, h_B, h_C, rows, cols);
    end = clock();
    double cpuTime = ((double) (end - start)) / CLOCKS_PER_SEC;

    // allocate device mem
    float *d_A, *d_B, *d_C;
    CHECK(hipMalloc((float**)&d_A, nBytes));
    CHECK(hipMalloc((float**)&d_B, nBytes));
    CHECK(hipMalloc((float**)&d_C, nBytes));

    // copy data from CPU to GPU
    start = clock();
    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));
    end = clock();
    double copyTime = ((double) (end - start)) / CLOCKS_PER_SEC;

    // launch CUDA kernel
    dim3 block(cols);
    dim3 grid((rows*cols+block.x-1)/block.x);
    start = clock();
    sumArraysOnDevice<<<grid, block>>>(d_A, d_B, d_C);
    CHECK(hipDeviceSynchronize()); // synchronize kernel only for debugging!
    end = clock();
    double gpuTime = ((double) (end - start)) / CLOCKS_PER_SEC;
    
    // copy data from GPU back to CPU
    CHECK(hipMemcpy(h_C_gpu, d_C, nBytes, hipMemcpyDeviceToHost));

    // verify
    verifyResult(h_C, h_C_gpu, rows, cols);
    printf("It takes %.2f sec to execute on CPU\n", cpuTime);
    printf("It takes %.2f sec to copy data from CPU to GPU\n", copyTime);
    printf("It takes %.2f sec to execute on GPU\n", gpuTime);

    // free host mem
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C_gpu);

    // free device mem
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    // clean up all resources
    hipDeviceReset();

    return 0;
}

/**********CUDA kernels**********/

__global__ void sumArraysOnDevice(float *A, float *B, float *C) {
    // 1D grid of 1D block
    // compute global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    C[idx] = A[idx] + B[idx];
}

/**********host functions**********/

void initialData(float *ip, int rows, int cols) {
    // generate different seed for random number
    time_t t;
    srand((unsigned int) time(&t));

    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            int idx = i * cols + j;
            ip[idx] = (float) (rand() & 0xFF) / 10.f;
        }
    }
}

void sumArraysOnHost(float *A, float *B, float *C, const int rows, const int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            int idx = i * cols + j;
            C[idx] = A[idx] + B[idx];
        }
    }
}

void verifyResult(float *hostRes, float *deviceRes, const int rows, const int cols) {
    double eps = 1e-8;
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            int idx = i * cols + j;
            if (abs(hostRes[idx] - deviceRes[idx] > eps)) {
                printf("Arrays do not match! Check your kernel code!\n");
                printf("host %5.2f gpu %5.2f at (%d, %d)\n", hostRes[idx], deviceRes[idx], i, j);
                return;
            }
        }
    }
    printf("Arrays match! Congrats, your kernel code works well!\n");
    return;
}